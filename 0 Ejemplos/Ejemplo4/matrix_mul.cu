#include <stdio.h>
#include "hipblas.h"
#include "matrix_mul.h"

// Host multiplication function
// Compute C = A * B
// hA is the height of A
// wA is the width of A
// wB is the width of B

extern "C"
void Mul(float* A, float* B, int hA, int wA, int wB,
	float* C)
{
	int size;

	// Load A and B to the device
	float* Ad;
	size = hA * wA * sizeof(float);
	hipMalloc((void**)&Ad, size);
	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
	float* Bd;
	size = wA * wB * sizeof(float);
	hipMalloc((void**)&Bd, size);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

	// Allocate C on the device
	float* Cd;
	size = hA * wB * sizeof(float);
	hipMalloc((void**)&Cd, size);

	// Compute the execution configuration
	float alpha = 1.0, beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(handle,
		HIPBLAS_OP_N, HIPBLAS_OP_N,
		hA,				/* [m] */ 
		wA,				/* [n] */  
		wB,				/* [k] */ 
		&alpha,				/* alfa */ 
		Ad, wA,			/* A[m][k], num columnas (lda) */ 
		Bd, wB,			/* B[k][n], num columnas (ldb) */
		&beta,				/* beta */
		C, wB			/* C[m][n], num columnas (ldc) */
	);
	hipblasDestroy(handle);

	// Read C from the device
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
}
