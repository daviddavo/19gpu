#include <stdio.h>
#include "hipblas.h"
#include "matrix_mul.h"

// Host multiplication function
// Compute C = A * B
// hA is the height of A
// wA is the width of A
// wB is the width of B

#define CUDACHECK(f) { \
    const hipError_t error = f; \
    if (error != hipSuccess) { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

#define CUBLASCHECK(f) { \
    const hipblasStatus_t error = f; \
    if (error != HIPBLAS_STATUS_SUCCESS) { \
        printf("Error %s:%d, ", __FILE__, __LINE__); \
        printf("code: %d\n", error); \
        exit(1); \
    } \
}

extern "C"
void Mul(float* A, float* B, int hA, int wA, int wB,
	float* C)
{
	int size;

	// Load A and B to the device
	float* Ad;
	size = hA * wA * sizeof(float);
	CUDACHECK(hipMalloc((void**)&Ad, size));
	CUDACHECK(hipMemcpy(Ad, A, size, hipMemcpyHostToDevice));
	float* Bd;
	size = wA * wB * sizeof(float);
	CUDACHECK(hipMalloc((void**)&Bd, size));
	CUDACHECK(hipMemcpy(Bd, B, size, hipMemcpyHostToDevice));

	// Allocate C on the device
	float* Cd;
	size = hA * wB * sizeof(float);
	CUDACHECK(hipMalloc((void**)&Cd, size));

	// Compute the execution configuration
	const float alpha = 1.0f, beta = 0.0f;
	hipblasHandle_t handle;
	CUBLASCHECK(hipblasCreate(&handle));
    printf("m: %d, n: %d, k: %d\n", hA, wB, wA);
	CUBLASCHECK(hipblasSgemm(handle,
		HIPBLAS_OP_N, HIPBLAS_OP_N,
		hA,				/* [m] */ 
		wB,				/* [n] */  
		wA,				/* [k] */ 
		&alpha,				/* alfa */ 
		Ad, wA,			/* A[m][k], num columnas (lda) */ 
		Bd, wB,			/* B[k][n], num columnas (ldb) */
		&beta,				/* beta */
		Cd, wB			/* C[m][n], num columnas (ldc) */
	));
	CUBLASCHECK(hipblasDestroy(handle));

	// Read C from the device
	CUDACHECK(hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost));

	// Free device memory
	CUDACHECK(hipFree(Ad));
	CUDACHECK(hipFree(Bd));
	CUDACHECK(hipFree(Cd));
}
